#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <ctime>

#define Nglobal 4096

__global__ void add(int *rowP, int *rowC, int i) 
{ 
	if(blockIdx.x == 0 || blockIdx.x == i-1){
		rowC[blockIdx.x] = 1;
	}
	else{
		rowC[blockIdx.x] = rowP[blockIdx.x-1] + rowP[blockIdx.x];
	}
} 

hipError_t GenerateTriangle(int n){
	int previousRow[Nglobal+1];
	for(int i = 0; i <= n; i++){
		previousRow[i] = 0;
	}
	previousRow[0] = 1;
	previousRow[1] = 1;
	int currentRow[Nglobal+1];
	for(int i = 0; i <= n; i++){
		currentRow[i] = 0;
	}
	int *dPreviousRow, *dCurrentRow;
	hipMalloc((void **) &dPreviousRow, sizeof(previousRow));
	hipMalloc((void **) &dCurrentRow, sizeof(currentRow));
	hipMemcpy(dPreviousRow, previousRow, sizeof (previousRow), hipMemcpyHostToDevice);
	hipMemcpy(dCurrentRow, currentRow, sizeof (currentRow), hipMemcpyHostToDevice);
	for(int i = 3; i <= n+1; i++){
		//actual calculations are done in this loop
		add<<<i,1>>>(dPreviousRow, dCurrentRow, i);
		hipMemcpy(dPreviousRow, dCurrentRow, (n+1)* sizeof (int), hipMemcpyDeviceToDevice);
	}
	//at this point currentRow needs to be copied to finRow
	hipMemcpy(currentRow, dCurrentRow, (n+1)* sizeof (int), hipMemcpyDeviceToHost);
	for(int i = 0; i < n+1; i++){
		printf("%d ",currentRow[i]);
	}
	printf("\n");
	return hipSuccess;
}

int main( int argc, const char* argv[] )
{
	clock_t Start = clock();
	int n = Nglobal;
	printf( "N is %d\n", n);
	if(n==1){
		printf("1\n");
		return 0;
	}
	if(n==2){
		printf("1 1\n");
		return 0;
	}
	hipError_t cudaStatus;
	cudaStatus = GenerateTriangle(n);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "generation failed!");
        return 1;
     }
	cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
     }
	int stopTime = clock();
	printf("Elapsed Time: %i Milliseconds\n",clock() - Start);
     return 0;
}